#include "include/commons.h"
#include "include/coo.h"
#include "include/csr.h"
#include "include/debug.h"
#include "include/kernels.cuh"
#include <hip/hip_runtime.h>
#include <hipsparse.h>


int testing();
int cuda_transpose_example();
// cuSPARSE Transpose CSR
int cuSparseCSRt(csr_matrix* in, csr_matrix* out) {
    printf("cuSparseCSRt\n");
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    size_t bufferSize = 0;
    int* d_in_row_offsets, *d_in_cols, *d_out_row_offsets, *d_out_cols;
    float* d_in_values, *d_out_values;
    // ? Allocate memory on device for Input Matrix
    printf("Now allocating %d bytes...\n", (in->rows + 1) * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_in_row_offsets, (in->rows + 1) * sizeof(int)));
    printf("Now allocating %d bytes...\n", in->nnz * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_in_cols, in->nnz * sizeof(int)));
    printf("Now allocating %d bytes...\n", in->nnz * sizeof(float));
    CHECK_CUDA(hipMalloc((void**)&d_in_values, in->nnz * sizeof(float)));
    // ? Allocate memory on device for Output Matrix
    printf("Now allocating %d bytes...\n", (out->rows + 1) * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_out_row_offsets, (out->rows + 1) * sizeof(int)));
    printf("Now allocating %d bytes...\n", out->nnz * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_out_cols, out->nnz * sizeof(int)));
    printf("Now allocating %d bytes...\n", out->nnz * sizeof(float));
    CHECK_CUDA(hipMalloc((void**)&d_out_values, out->nnz * sizeof(float)));
    // ? Copy data from host to device for Input Matrix
    CHECK_CUDA(hipMemcpy(d_in_row_offsets, in->row_offsets, (in->rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_cols, in->col_indices, in->nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_values, in->values, in->nnz * sizeof(float), hipMemcpyHostToDevice));
    // ? Find buffer size to perform the transpose
    hipsparseCsr2cscEx2_bufferSize(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT, &bufferSize
    );
    printf("Buffer size: %ld\n", bufferSize);
    void* dBuffer = NULL;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    // ? Perform the actual transpose operation on device
    hipsparseCsr2cscEx2(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT, dBuffer
    );
    // ? Copy data from device to host for Output Matrix
    CHECK_CUDA(hipMemcpy(out->row_offsets, d_out_row_offsets, out->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->col_indices, d_out_cols, out->nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->values, d_out_values, out->nnz * sizeof(float), hipMemcpyDeviceToHost));

    // ? Free memory on device
    CHECK_CUDA(hipFree(d_in_row_offsets));
    CHECK_CUDA(hipFree(d_in_cols));
    CHECK_CUDA(hipFree(d_in_values));
    CHECK_CUDA(hipFree(d_out_row_offsets));
    CHECK_CUDA(hipFree(d_out_cols));
    CHECK_CUDA(hipFree(d_out_values));
    CHECK_CUDA(hipFree(dBuffer));

    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
    return 0;
}

int main(int argc, char** argv) {
#ifdef DEBUG
    if (argc > 1) {
        printf("argc = %d:\n", argc);
        for (int i = 0; i < argc; i++) {
            printf("arg %d : %s\n", i+1, argv[i]);
        }
        printf("\n");
    }
#endif
    //try coo kernel
    //cuda_transpose_example();

    csr_matrix* csr = load_csr_matrix("matrices/tests/mockcsr.mtx");
    csr_matrix* csr_t = new csr_matrix(csr->rows, csr->cols, csr->nnz);
    //pretty_print_csr_matrix(csr);
    cuSparseCSRt(csr, csr_t);
    //pretty_print_csr_matrix(csr_t);
    if (is_transpose(csr, csr_t)) {
        printf("Transpose is correct\n");
    } else {
        printf("Transpose is incorrect\n");
    }
    delete csr;
    delete csr_t;

    return 0;
}

int cuda_transpose_example() {
    coo_matrix* coo = load_coo_matrix("matrices/circuit204.mtx");
    coo_element* el = coo->el;
    coo_matrix* d_coo;
    coo_element* d_el;
    hipMallocManaged((void**)&d_coo, sizeof(coo_matrix));
    hipMallocManaged((void**)&d_el, coo->nnz * sizeof(coo_element));
    hipMemcpy(d_coo, coo, sizeof(coo_matrix), hipMemcpyHostToDevice);
    hipMemcpy(d_el, el, coo->nnz * sizeof(coo_element), hipMemcpyHostToDevice);
    PRINTF("Copied memory\n");
    d_coo->el = d_el;
    printf("Before transpose\n");
    print_coo_less(d_coo);
    coo_transpose<<<coo->nnz,1>>>(d_coo);
    hipMemcpy(d_coo, d_coo, sizeof(coo_matrix), hipMemcpyDeviceToHost);
    printf("After transpose\n");
    print_coo_less(d_coo);

    hipFree(d_coo);
    hipFree(d_el);
    delete[] coo->el;
    delete coo;
    return 0;
}

int testing() {
    coo_matrix* coo = load_coo_matrix("matrices/tests/mockcoo.mtx");
    PRINTF("--------------------\n");
    print_coo_matrix(coo);
    PRINTF("--------------------\n");
    delete coo;
    coo = load_coo_matrix("matrices/circuit204.mtx");
    print_coo_metadata(coo);
    int full_size = 4;
    float *mat = new float [full_size*full_size];
    sparseInitMatrix(mat, full_size);
    coo_matrix* coo2 = mat_to_coo(mat, full_size);
    PRINTF("--------------------\n");
    print_coo_matrix(coo2);
    printMatrix(mat, full_size);
    PRINTF("--------------------\n");

    delete[] mat;
    delete coo;
    delete coo2;

    PRINTF("CSR tests\n");
    //csr_matrix* csr = load_csr_matrix("matrices/tests/mockcsr.mtx");
    csr_matrix* csr = load_csr_matrix();
    PRINTF("--------------------\n");
    print_csr_matrix(csr);
    PRINTF("--------------------\n");
    return 0;
}
