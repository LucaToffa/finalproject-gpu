#include "hip/hip_runtime.h"
#include "../include/coo.h"
#include "../include/kernels.cuh"
#include "../include/commons.h"
#include <hip/hip_runtime.h>
#include <cassert>

__global__ void dummy_kernel() {
    return;
}
/**
 * @brief Kernel to Transpose a COO Matrix out of place
    * @param[in] in - COO Matrix to be transposed
    * @param[out] out - Transposed COO Matrix
 */
__global__ void cuCOOtCopy(coo_element *in, coo_element *out, size_t nnz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < nnz) {
        out[i].row = in[i].col;
        out[i].col = in[i].row;
        out[i].val = in[i].val;
    }
}
/**
 * @brief Kernel to Transpose a COO Matrix in-place
    * @param[in] in - COO Matrix to be transposed
 */
__global__ void cuCOOt(coo_element *in, size_t nnz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < nnz) {
        size_t tmp = in[i].row;
        in[i].row = in[i].col;
        in[i].col = tmp;
    }
}

// Kernel to count the number of non-zero entries per column
__global__ void countNNZPerColumn(const int* col_indices, int* col_counts, int nnz) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < nnz) {
        atomicAdd(&col_counts[col_indices[tid]], 1);
    }
}

// // Kernel to scatter values and row indices to transposed matrix
// __global__ void scatterToTransposed(const float* values, const int* col_indices, const int* row_ptr,
//                                     float* t_values, int* t_row_indices, int* t_col_ptr, int num_rows) {
//     int row = blockIdx.x * blockDim.x + threadIdx.x;
//     if (row < num_rows) {
//         for (int j = row_ptr[row]; j < row_ptr[row + 1]; ++j) {
//             int col = col_indices[j];
//             int dest = atomicAdd(&t_col_ptr[col], 1);
//             t_values[dest] = values[j];
//             t_row_indices[dest] = row;
//         }
//     }
// }

//1 thread per col, .append if col == thdx
//join the threads in order
__global__ void order_by_column(const float* values, const int* col_indices, //col_offset
                                float* d_t_values, int* t_col_indices, int *d_col_counts,
                                int num_cols, int nnz,
                                int *d_t_col_indices, int *d_t_col_indices_ordered){
    int col = threadIdx.x + blockIdx.x * blockDim.x; //current working column
    //how many values are in this column?
    int start_offset = t_col_indices[col]; //col_ptr 0 1 2 4 7 
    //int num_values = d_col_counts[col]; //1 1 2 3
    int pos = 0;
    if (col < num_cols) {
        for(int i = 0; i < nnz; i++){
            if(col == col_indices[i]){
                //append to the end of the array val
                d_t_values[start_offset + pos] = values[i]; 
                d_t_col_indices_ordered[start_offset + pos] = d_t_col_indices[i];
                pos++;
            }
        }
    }
}
/**
    01 __ global__ void transp (int *AT.idx, ……) {
    02   tid = blockIdx.x * blockDim.x + threadIdx.x ;
    03   while (tid < NNZ) {
    04        temp = csr_t->row_ptr[csr->col_indeces[tid]] + tex_off[tid];
    05        csr_t->col_indeces[temp] = idxtemp[tid];
    06        csr_t->values[temp] = csr->values[tid];
    07        tid + = THREADS;
    08   }
    09 }
 */
__global__ void csr_matrix_transpose_kernel(
    const int num_rows,       // Number of rows in original matrix
    const int num_cols,       // Number of columns in original matrix
    const int nnz,            // Number of non-zero elements

    // Input CSR matrix components
    int* col_indices, // Column indices array input
    float* values,    // Values array input

    // Output transposed CSR matrix components
    int* t_row_ptr,         // Transposed row pointer array
    int* t_col_indices,     // Transposed column indices array
    float* t_values,         // Transposed values array

    int* tex_off,
    int* idxtemp
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < nnz) {
        int temp = t_row_ptr[col_indices[tid]] + tex_off[tid];
        t_col_indices[temp] = idxtemp[tid];
        t_values[temp] = values[tid];
        tid += blockDim.x * gridDim.x;
    }
    __syncthreads();
}

//old version of block transpose algorithm to check against the new ones
__global__ void block_transpose(float *input, float *output, int N){
    __shared__ float tile[TILE_SIZE][TILE_SIZE+1];
    
    //input to shared offsets
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;
    int w = gridDim.x * TILE_SIZE;

    for(int i = 0; i < TILE_SIZE; i += BLOCK_ROWS){
        tile[threadIdx.y+i][threadIdx.x] = input[(y+i) * w + x];
    }

    __syncthreads();

    //shared to output offsets
    x = blockIdx.y * TILE_SIZE + threadIdx.x;
    y = blockIdx.x * TILE_SIZE + threadIdx.y;

    for(int j = 0; j < TILE_SIZE; j += BLOCK_ROWS){
        output[(y+j) * w + x] = tile[threadIdx.x][threadIdx.y+j];
    }
}

//without the +1 the memory access conflicts cannot be avoided
__global__ void conflict_transpose(float *input, float *output){
    __shared__ float tile[TILE_SIZE][TILE_SIZE];
    
    //input to shared offsets
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;
    int w = gridDim.x * TILE_SIZE;

    for(int i = 0; i < TILE_SIZE; i += BLOCK_ROWS){
        tile[threadIdx.y+i][threadIdx.x] = input[(y+i) * w + x];
    }

    __syncthreads();

    //shared to output offsets
    x = blockIdx.y * TILE_SIZE + threadIdx.x;
    y = blockIdx.x * TILE_SIZE + threadIdx.y;

    for(int j = 0; j < TILE_SIZE; j += BLOCK_ROWS){
        output[(y+j) * w + x] = tile[threadIdx.x][threadIdx.y+j];
    }

}

#define B_TILE TILE_SIZE
#define B_ROWS BLOCK_ROWS
// implementation of basic transpose in gpu
// to avoid ifs in the kernel, check the matrix size and derive block/threads size
__global__ void basic_transpose(float *input, float *output, int N){
    //matrix transpose that works for any size
    int x = blockIdx.x * B_TILE + threadIdx.x;
    int y = blockIdx.y * B_TILE + threadIdx.y;

    int index_in = x + N * y;
    int index_out = y + N * x;

    for (int i = 0; i < B_TILE; i += B_ROWS){
        output[index_out + i] = input[index_in + i * N];
    }
}

__global__ void prefix_scan(int *g_odata, int *g_idata, int n, int *last)
{   
    extern __shared__ int temp[]; // allocated on invocation
    int thid = threadIdx.x;// + blockIdx.x * blockDim.x;
    // if(thid >= 2*n) return;
    //printf("prefix_scan) n: %d, thr: %d\n", n, thid);
    int offset = 1;
    temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
    temp[2 * thid + 1] = g_idata[2 * thid + 1];
    //printf("prefix_scan) temp[2 * thid]: %d, temp[2 * thid + 1]: %d\n", temp[2 * thid], temp[2 * thid + 1]);
    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        //printf("prefix_scan) d: %d in thr: %d\n", d, thid);
        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    if (thid == 0)
    {
        //printf("prefix_scan) d: 0 in thr: %d\n", thid);
        last[0] = temp[n - 1]; // write the last element of the scan to the last element of the block
        temp[n - 1] = 0;
    }                              // clear the last element
    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
    g_odata[2 * thid + 1] = temp[2 * thid + 1];
}

