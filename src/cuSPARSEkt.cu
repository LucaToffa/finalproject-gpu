#include "../include/commons.h"
#include "../include/csr.h"
#include "../include/cuSPARSEkt.cuh"
#include <hip/hip_runtime.h>
#include <hipsparse.h>


int cuSparseCSRt(csr_matrix* in, csr_matrix* out) {
    printf("cuSparseCSRt\n");
    // ? Create cuSPARSE handle and matrix descriptor
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    // ? Set matrix type and index base
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    size_t bufferSize = 0;
    int* d_in_row_offsets, *d_in_cols, *d_out_row_offsets, *d_out_cols;
    float* d_in_values, *d_out_values;
    // ? Allocate memory on device for Input Matrix
    printf("Now allocating %lu bytes...\n", (unsigned long)(in->rows + 1) * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_in_row_offsets, (in->rows + 1) * sizeof(int)));
    printf("Now allocating %lu bytes...\n", (unsigned long)in->nnz * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_in_cols, in->nnz * sizeof(int)));
    printf("Now allocating %lu bytes...\n", (unsigned long)in->nnz * sizeof(float));
    CHECK_CUDA(hipMalloc((void**)&d_in_values, in->nnz * sizeof(float)));
    // ? Allocate memory on device for Output Matrix
    printf("Now allocating %lu bytes...\n", (unsigned long)(out->rows + 1) * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_out_row_offsets, (out->rows + 1) * sizeof(int)));
    printf("Now allocating %lu bytes...\n", (unsigned long)out->nnz * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_out_cols, out->nnz * sizeof(int)));
    printf("Now allocating %lu bytes...\n", (unsigned long)out->nnz * sizeof(float));
    CHECK_CUDA(hipMalloc((void**)&d_out_values, out->nnz * sizeof(float)));
    // ? Create cuda events to measure time
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    // ? Copy data from host to device for Input Matrix
    CHECK_CUDA(hipMemcpy(d_in_row_offsets, in->row_offsets, (in->rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_cols, in->col_indices, in->nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_values, in->values, in->nnz * sizeof(float), hipMemcpyHostToDevice));
    // ? Find buffer size to perform the transpose
    hipsparseCsr2cscEx2_bufferSize(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &bufferSize
    );
    printf("Buffer size: %lu\n", bufferSize);
    // ? Allocate memory on device for buffer
    void* dBuffer = NULL;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    // ? Record time before performing the transpose operation
    CHECK_CUDA(hipEventRecord(start));
    // ? Perform the actual transpose operation on device
    hipsparseCsr2cscEx2(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, dBuffer
    );
    // ? Record time after performing the transpose operation
    CHECK_CUDA(hipEventRecord(stop));
    // ? Copy data from device to host for Output Matrix
    CHECK_CUDA(hipMemcpy(out->row_offsets, d_out_row_offsets, out->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->col_indices, d_out_cols, out->nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->values, d_out_values, out->nnz * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for executing cuSPARSECSRt operation: %f ms\n", milliseconds);
    //TODO: correct output
    // ? Free memory on device
    CHECK_CUDA(hipFree(d_in_row_offsets));
    CHECK_CUDA(hipFree(d_in_cols));
    CHECK_CUDA(hipFree(d_in_values));
    CHECK_CUDA(hipFree(d_out_row_offsets));
    CHECK_CUDA(hipFree(d_out_cols));
    CHECK_CUDA(hipFree(d_out_values));
    CHECK_CUDA(hipFree(dBuffer));
    // ? Destroy cuSPARSE handle and matrix descriptor
    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
    return 0;
}
