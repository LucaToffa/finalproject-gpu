#include "hip/hip_runtime.h"
#include "../include/commons.h"
#include "../include/debug.h"
#include "../include/csr.h"
#include "../include/cuSPARSEkt.cuh"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <fstream>
#include "../include/kernels.cuh"


int cuSparseCSRt(const csr_matrix* in, csr_matrix* out, int matrix_size) {
    assert(in != NULL && out != NULL && in->row_offsets != NULL && in->col_indices != NULL && in->values != NULL && out->row_offsets != NULL && out->col_indices != NULL && out->values != NULL);
    assert(in->rows == out->cols);
    assert(in->cols == out->rows);
    if(hipSetDevice(0) != hipSuccess) {
        fprintf(stderr, "Failed to set CUDA device\n");
        return 1;
    }
    PRINTF("cuSparseCSRt\n");

    // ? Create cuSPARSE handle and matrix descriptor
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    // ? Create cuda events to measure time
    hipEvent_t start, stop, startK, stopK;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventCreate(&startK));
    CHECK_CUDA(hipEventCreate(&stopK));

    dummy_kernel<<<1,1>>>(); // ? Warm up the GPU

    CHECK_CUDA(hipEventRecord(start));

    size_t bufferSize = 0;
    int* d_in_row_offsets, *d_in_cols, *d_out_row_offsets, *d_out_cols;
    float* d_in_values, *d_out_values;
    // ? Allocate memory on device for Input Matrix
    CHECK_CUDA(hipMalloc((void**)&d_in_row_offsets, (in->rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_in_cols, in->nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_in_values, in->nnz * sizeof(float)));
    // ? Allocate memory on device for Output Matrix
    CHECK_CUDA(hipMalloc((void**)&d_out_row_offsets, (in->cols + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_out_cols, out->nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_out_values, out->nnz * sizeof(float)));
   
    // ? Copy data from host to device for Input Matrix
    CHECK_CUDA(hipMemcpy(d_in_row_offsets, in->row_offsets, (in->rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_cols, in->col_indices, in->nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_values, in->values, in->nnz * sizeof(float), hipMemcpyHostToDevice));

    void* dBuffer = NULL;

    CHECK_CUDA(hipEventRecord(startK))
    for(int i = 0; i < TRANSPOSITIONS; i++) {
        // ? Find buffer size to perform the transpose
        CHECK_CUSPARSE(hipsparseCsr2cscEx2_bufferSize(
            handle,
            in->rows, in->cols, in->nnz,
            d_in_values, d_in_row_offsets, d_in_cols,
            d_out_values, d_out_row_offsets, d_out_cols,
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &bufferSize
        ));
        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize)); // ? Allocate memory on device for buffer
        // ? Record time before performing the transpose operation
        // CHECK_CUDA(hipEventRecord(start));
        // ? Perform the actual transpose operation on device
        CHECK_CUSPARSE(hipsparseCsr2cscEx2(
            handle,
            in->rows, in->cols, in->nnz,
            d_in_values, d_in_row_offsets, d_in_cols,
            d_out_values, d_out_row_offsets, d_out_cols,
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, dBuffer
        ));
        CHECK_CUDA(hipFree(dBuffer));
    }
    // ? Record time after performing the transpose operation
    CHECK_CUDA(hipEventRecord(stopK));
    // ? Copy data from device to host for Output Matrix
    CHECK_CUDA(hipMemcpy(out->row_offsets, d_out_row_offsets, (in->rows + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->col_indices, d_out_cols, out->nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->values, d_out_values, out->nnz * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipEventRecord(stop));

    CHECK_CUDA(hipEventSynchronize(stopK));
    CHECK_CUDA(hipEventSynchronize(stop));

    CHECK_CUDA(hipDeviceSynchronize());

    float millisecondsK = 0;
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&millisecondsK, startK, stopK));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    int N = in->cols; /* *** should be the real matrix size */
    float ogbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / milliseconds;
    float kgbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / millisecondsK;
    millisecondsK /= (float)TRANSPOSITIONS;
    milliseconds /= (float)TRANSPOSITIONS;
    PRINTF("Time for executing cuSPARSECSRt operation: %f ms\n", milliseconds);
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    PRINTF("Operation Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Kernel Throughput in GB/s: %7.2f\n", kgbs);

    std::ofstream output;
    output.open("logs/results.log", std::ios::out | std::ios_base::app);
    //output << "Cusparse, " << "OpTime, Op-GB/s, " << milliseconds << "K-GB/s\n";
    // algorithm, OpTime, Op-GB/s, KTime, K-GB/s
    output << "CUsparse, " << matrix_size << "x" << matrix_size << ", " << milliseconds << ", "<< ogbs << ", " << millisecondsK << ", " << kgbs << "\n"; /* *** */
    output.close();

    // Check if transpose was successful
    PRINTF("Now checking if transpose was successful\n");
    if (is_transpose(in, out)) {
        PRINTF("Transpose is correct\n");
    } else {
        printf("Transpose is incorrect\n");
        // save to log file
        std::ofstream logstream;
        logstream.open("logs/cusparse_transpose_err.log", std::ios::out);
        logstream << "Original Matrix:\n";
        pretty_print_csr_matrix(in, logstream);
        logstream << "\n\nTransposed Matrix:\n";
        pretty_print_csr_matrix(out, logstream);
        logstream.close();
    }

    //TODO: correct output
    // ? Free memory on device
    CHECK_CUDA(hipFree(d_in_row_offsets));
    CHECK_CUDA(hipFree(d_in_cols));
    CHECK_CUDA(hipFree(d_in_values));
    CHECK_CUDA(hipFree(d_out_row_offsets));
    CHECK_CUDA(hipFree(d_out_cols));
    CHECK_CUDA(hipFree(d_out_values));
    // CHECK_CUDA(hipFree(dBuffer));
    // ? Destroy cuSPARSE handle and matrix descriptor
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    return 0;
}
