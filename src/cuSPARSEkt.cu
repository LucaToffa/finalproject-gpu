#include "../include/commons.h"
#include "../include/csr.h"
#include "../include/cuSPARSEkt.cuh"
#include <hip/hip_runtime.h>
#include <hipsparse.h>


int cuSparseCSRt(csr_matrix* in, csr_matrix* out) {
    printf("cuSparseCSRt\n");
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    size_t bufferSize = 0;
    int* d_in_row_offsets, *d_in_cols, *d_out_row_offsets, *d_out_cols;
    float* d_in_values, *d_out_values;
    // ? Allocate memory on device for Input Matrix
    printf("Now allocating %d bytes...\n", (in->rows + 1) * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_in_row_offsets, (in->rows + 1) * sizeof(int)));
    printf("Now allocating %d bytes...\n", in->nnz * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_in_cols, in->nnz * sizeof(int)));
    printf("Now allocating %d bytes...\n", in->nnz * sizeof(float));
    CHECK_CUDA(hipMalloc((void**)&d_in_values, in->nnz * sizeof(float)));
    // ? Allocate memory on device for Output Matrix
    printf("Now allocating %d bytes...\n", (out->rows + 1) * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_out_row_offsets, (out->rows + 1) * sizeof(int)));
    printf("Now allocating %d bytes...\n", out->nnz * sizeof(int));
    CHECK_CUDA(hipMalloc((void**)&d_out_cols, out->nnz * sizeof(int)));
    printf("Now allocating %d bytes...\n", out->nnz * sizeof(float));
    CHECK_CUDA(hipMalloc((void**)&d_out_values, out->nnz * sizeof(float)));
    // ? Copy data from host to device for Input Matrix
    CHECK_CUDA(hipMemcpy(d_in_row_offsets, in->row_offsets, (in->rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_cols, in->col_indices, in->nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_values, in->values, in->nnz * sizeof(float), hipMemcpyHostToDevice));
    // ? Find buffer size to perform the transpose
    hipsparseCsr2cscEx2_bufferSize(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT, &bufferSize
    );
    printf("Buffer size: %ld\n", bufferSize);
    void* dBuffer = NULL;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    // ? Perform the actual transpose operation on device
    hipsparseCsr2cscEx2(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT, dBuffer
    );
    // ? Copy data from device to host for Output Matrix
    CHECK_CUDA(hipMemcpy(out->row_offsets, d_out_row_offsets, out->rows * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->col_indices, d_out_cols, out->nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->values, d_out_values, out->nnz * sizeof(float), hipMemcpyDeviceToHost));

    // ? Free memory on device
    CHECK_CUDA(hipFree(d_in_row_offsets));
    CHECK_CUDA(hipFree(d_in_cols));
    CHECK_CUDA(hipFree(d_in_values));
    CHECK_CUDA(hipFree(d_out_row_offsets));
    CHECK_CUDA(hipFree(d_out_cols));
    CHECK_CUDA(hipFree(d_out_values));
    CHECK_CUDA(hipFree(dBuffer));

    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
    return 0;
}
