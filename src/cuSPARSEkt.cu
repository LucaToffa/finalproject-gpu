#include "hip/hip_runtime.h"
#include "../include/commons.h"
#include "../include/csr.h"
#include "../include/cuSPARSEkt.cuh"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <fstream>


int cuSparseCSRt(const csr_matrix* in, csr_matrix* out) {
    assert(in != NULL && out != NULL && in->row_offsets != NULL && in->col_indices != NULL && in->values != NULL && out->row_offsets != NULL && out->col_indices != NULL && out->values != NULL);
    assert(in->rows == out->cols);
    assert(in->cols == out->rows);
    if(hipSetDevice(0) != hipSuccess) {
        fprintf(stderr, "Failed to set CUDA device\n");
        return 1;
    }
    printf("cuSparseCSRt\n");
    // ? Create cuSPARSE handle and matrix descriptor
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    // ? Set matrix type and index base
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    size_t bufferSize = 0;
    int* d_in_row_offsets, *d_in_cols, *d_out_row_offsets, *d_out_cols;
    float* d_in_values, *d_out_values;
    // ? Allocate memory on device for Input Matrix
    CHECK_CUDA(hipMalloc((void**)&d_in_row_offsets, (in->rows + 1) * sizeof(size_t)));
    CHECK_CUDA(hipMalloc((void**)&d_in_cols, in->nnz * sizeof(size_t)));
    CHECK_CUDA(hipMalloc((void**)&d_in_values, in->nnz * sizeof(float)));
    // ? Allocate memory on device for Output Matrix
    CHECK_CUDA(hipMalloc((void**)&d_out_row_offsets, (out->cols + 1) * sizeof(size_t)));
    CHECK_CUDA(hipMalloc((void**)&d_out_cols, out->nnz * sizeof(size_t)));
    CHECK_CUDA(hipMalloc((void**)&d_out_values, out->nnz * sizeof(float)));
    // ? Create cuda events to measure time
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    // ? Copy data from host to device for Input Matrix
    CHECK_CUDA(hipMemcpy(d_in_row_offsets, in->row_offsets, (in->rows + 1) * sizeof(size_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_cols, in->col_indices, in->nnz * sizeof(size_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_in_values, in->values, in->nnz * sizeof(float), hipMemcpyHostToDevice));
    // ? Find buffer size to perform the transpose
    hipsparseCsr2cscEx2_bufferSize(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &bufferSize
    );
    printf("Buffer size: %lu\n", bufferSize);
    // ? Allocate memory on device for buffer
    void* dBuffer = NULL;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    // ? Record time before performing the transpose operation
    CHECK_CUDA(hipEventRecord(start));
    // ? Perform the actual transpose operation on device
    hipsparseCsr2cscEx2(
        handle,
        in->rows, in->cols, in->nnz,
        d_in_values, d_in_row_offsets, d_in_cols,
        d_out_values, d_out_row_offsets, d_out_cols,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, dBuffer
    );
    // ? Record time after performing the transpose operation
    CHECK_CUDA(hipEventRecord(stop));
    // ? Copy data from device to host for Output Matrix
    CHECK_CUDA(hipMemcpy(out->row_offsets, d_out_row_offsets, (out->cols + 1) * sizeof(size_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->col_indices, d_out_cols, out->nnz * sizeof(size_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(out->values, d_out_values, out->nnz * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for executing cuSPARSECSRt operation: %f ms\n", milliseconds);

    std::ofstream output;
    output.open ("logs/results.log", std::ios::out | std::ios_base::app);
    output << "N_mat, " << "Cusparse, " << "OpTime, Op-GB/s, " << milliseconds << "K-GB/s\n";
    output.close();

    // Check if transpose was successful
    printf("Now checking if transpose was successful\n");
    if (is_transpose(in, out)) {
        printf("Transpose is correct\n");
    } else {
        printf("Transpose is incorrect\n");
        // save to log file
        std::ofstream logstream;
        logstream.open("logs/cusparse_transpose_err.log", std::ios::out);
        logstream << "Original Matrix:\n";
        pretty_print_csr_matrix(in, logstream);
        logstream << "\n\nTransposed Matrix:\n";
        pretty_print_csr_matrix(out, logstream);
        logstream.close();
    }

    //TODO: correct output
    // ? Free memory on device
    CHECK_CUDA(hipFree(d_in_row_offsets));
    CHECK_CUDA(hipFree(d_in_cols));
    CHECK_CUDA(hipFree(d_in_values));
    CHECK_CUDA(hipFree(d_out_row_offsets));
    CHECK_CUDA(hipFree(d_out_cols));
    CHECK_CUDA(hipFree(d_out_values));
    CHECK_CUDA(hipFree(dBuffer));
    // ? Destroy cuSPARSE handle and matrix descriptor
    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
    return 0;
}
