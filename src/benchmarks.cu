#include "hip/hip_runtime.h"
//complete run for each transposition algorithm callled by complete_benchmark in main
#include "../include/benchmarks.cuh"
#include "../include/commons.h"
#include "../include/kernels.cuh"
#include "../include/debug.h"
#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <fstream>

int coo_transposition(coo_matrix* coo, int matrix_size) {
    PRINTF("--------------------\n");
    PRINTF("COO Transposition Method called: coo_transposition() -> cuCOOt().\n");
    if ((hipSetDevice(0)) != hipSuccess) {
        printf("Failed to set CUDA device\n");
        return 1;
    }
    coo_element* el = coo->el;
    coo_matrix* d_coo;
    coo_element* d_el;

    hipEvent_t start, stop, startK, stopK;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventCreate(&startK));
    CHECK_CUDA(hipEventCreate(&stopK));

    dummy_kernel<<<1,1>>>(); // ? Warm up the GPU

    CHECK_CUDA(hipEventRecord(start));

    CHECK_CUDA(hipMallocManaged((void**)&d_coo, sizeof(coo_matrix)));
    CHECK_CUDA(hipMallocManaged((void**)&d_el, coo->nnz * sizeof(coo_element)));
    CHECK_CUDA(hipMemcpy(d_coo, coo, sizeof(coo_matrix), hipMemcpyHostToDevice));
    PRINTF("Copied & Allocated Memory Succesfully\n");
    d_coo->el = d_el;

    #ifdef DEBUG
        printf("Pre-Transpose Matrix:\n");
        print_coo_less(coo);
    #endif
    
    CHECK_CUDA(hipEventRecord(startK));
    for (int i = 0; i < TRANSPOSITIONS; i++) {
        hipMemcpy(d_el, el, coo->nnz * sizeof(coo_element), hipMemcpyHostToDevice);
        cuCOOt<<<(coo->nnz + 255) / 256, 256>>>(d_coo->el, d_coo->nnz);
    }
    //cuCOOt<<<coo->nnz,1>>>(d_coo->el, d_coo->nnz);
    CHECK_CUDA(hipEventRecord(stopK));

    CHECK_CUDA(hipMemcpy(d_coo, d_coo, sizeof(coo_matrix), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipEventRecord(stop));

    CHECK_CUDA(hipEventSynchronize(stopK));
    CHECK_CUDA(hipEventSynchronize(stop));

    CHECK_CUDA(hipDeviceSynchronize());

    int ret = 0;
    #ifdef DEBUG
        printf("Post-Transpose Matrix:\n");
        print_coo_less(d_coo);
    #endif
    if (is_transpose(coo, d_coo)) {
        PRINTF("Transpose is correct.\n");
    } else {
        ret = -1;
        printf("coo_transposition) Transpose is incorrect.\n");
    }

    float millisecondsK = 0;
    CHECK_CUDA(hipEventElapsedTime(&millisecondsK, startK, stopK));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    int N = coo->cols; 
    float ogbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / milliseconds;
    float kgbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / millisecondsK;
    milliseconds /= (float)TRANSPOSITIONS;
    millisecondsK /= (float)TRANSPOSITIONS;
    PRINTF("Time for executing cuCOOt operation: %f ms\n", milliseconds);
    PRINTF("Operation Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Kernel Throughput in GB/s: %7.2f\n", kgbs);

    std::ofstream output;
    output.open ("logs/results.log", std::ios::out | std::ios_base::app);
    // algorithm, OpTime, Op-GB/s, KTime, K-GB/s
    output << "COO, " << matrix_size << "x" << matrix_size << ", " <<  milliseconds << ", "<< ogbs << ", " << millisecondsK << ", " << kgbs << "\n"; /* *** */
    output.close();

    CHECK_CUDA(hipFree(d_coo));
    CHECK_CUDA(hipFree(d_el));
    PRINTF("Freed Memory Succesfully.\n");
    PRINTF("--------------------\n");
    return ret;
}

int csr_transposition_2(csr_matrix* csr, csr_matrix* csr_t, int matrix_size) {
    hipEvent_t start, stop;
    hipEvent_t startK, stopK;

    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventCreate(&startK));
    CHECK_CUDA(hipEventCreate(&stopK));
    dummy_kernel<<<1,1>>>(); // ? Warm up the GPU
    CHECK_CUDA(hipEventRecord(start));

    int *d_row_ptr, *d_col_indices;
    float *d_values;
    CHECK_CUDA(hipMalloc((void**)&d_row_ptr, (csr->rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_col_indices, csr->nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_values, csr->nnz * sizeof(float)));

    // CHECK_CUDA(hipMemcpy(d_row_ptr, csr->row_offsets, (csr->rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    // CHECK_CUDA(hipMemcpy(d_col_indices, csr->col_indices, csr->nnz * sizeof(int), hipMemcpyHostToDevice));
    // CHECK_CUDA(hipMemcpy(d_values, csr->values, csr->nnz * sizeof(float), hipMemcpyHostToDevice));

    int *d_t_row_ptr, *d_t_col_indices;
    float *d_t_values;
    CHECK_CUDA(hipMalloc((void**)&d_t_row_ptr, (csr->cols + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_t_col_indices, csr->nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_t_values, csr->nnz * sizeof(float)));

    int block_size = 64;
    int grid_size = std::min((csr->nnz + block_size - 1) / block_size, 1024);

    CHECK_CUDA(hipEventRecord(startK));
    //for(int i = 0; i < TRANSPOSITIONS; i++) {
        /**
        // construct auxiliary data arrays
        1 ∗intra = new int[nnz]();
        2 ∗inter = new int[(nthreads + 1) ∗ n]();
        3 ∗csrRowIdx = new int[nnz]();
        4 #pragma omp parallel for schedule(dynamic)
        5 for i ←0; i < m; i++ do
            6 for j ←csrRowPtr[i]; j <csrRowPtr[i+1]; j++ do
                7 csrRowIdx[j] = i;
        8 #pragma omp parallel
        9 // partition nnz evenly on threads, get start in csrColIdx
        and len for each thread
        10 for i ←0; i < len; i++ do
            11 intra[start + i]=inter[(tid + 1) ∗ n + csrColIdx[start + i]] + +;
        12 // vertical scan
        13 #pragma omp parallel for schedule(dynamic)
        14 for i ←0; i < n; i++ do
            15 for j ←1; j < nthread + 1; j++ do
                16 inter[i + n ∗ j]+ = inter[i + n ∗ (j − 1)];
        17 #pragma omp parallel for schedule(dynamic)
        18 for i ←0; i < n; i++ do
            19 cscColPtr[i + 1]=inter[n ∗ nthread + i];
        20 prefix sum(cscColP tr, n + 1);
        21 #pragma omp parallel
        22 for i ←0; i < len; i++ do
            23 loc = cscColPtr[csrColIdx[start + i]] + inter[tid ∗ n +
            csrColIdx[start + i]] + intra[start + i];
            24 cscRowIdx[loc] = csrRowIdx[start + i];
            25 cscVal[loc] = csrVal[start + i];
        26 // free intra, inter, csrRowIdx
        27 return;
         */

    // ? Implement everything in CPU for now
    int nthread = 0;
    int nthreads = 1;
    int* intra = new int[csr->nnz]();
    int* inter = new int[(nthreads + 1) * csr->cols]();
    int* csrRowIdx = new int[csr->nnz]();

    // ? Run this on GPU over i = thread_id = threadIdx.x + blockIdx.x * blockDim.x < csr->rows
    for(int i = 0; i < csr->rows; i++) {
        for(int j = csr->row_offsets[i]; j < csr->row_offsets[i+1]; j++) {
            csrRowIdx[j] = i;
        }
    }
    // ? Run this on GPU over i = thread_id = threadIdx.x + blockIdx.x * blockDim.x < csr->nnz
    for(int i = 0; i < csr->nnz; i++) {
        intra[i] = inter[(nthread + 1) * csr->col_indices[i]]++;
    }

    // ? Run this on GPU over i = thread_id = threadIdx.x + blockIdx.x * blockDim.x < csr->cols
    for(int i = 0; i < csr->cols; i++) {
        for(int j = 1; j < nthread + 1; j++) {
            inter[i + csr->cols * j] += inter[i + csr->cols * (j - 1)];
        }
    }

    for(int i = 0; i < csr->cols; i++) {
        csr_t->row_offsets[i + 1] = inter[csr->cols * (nthread) + i];
    }
    
    // prefix sum
    for(int i = 0; i < csr->cols; i++) {
        csr_t->row_offsets[i + 1] += csr_t->row_offsets[i];
    }

    for(int i = 0; i < csr->nnz; i++) {
        int loc = csr_t->row_offsets[csr->col_indices[i]] + inter[nthread * csr->cols + csr->col_indices[i]] + intra[i];
        csr_t->col_indices[loc] = csrRowIdx[i];
        csr_t->values[loc] = csr->values[i];
    }

    delete[] intra;
    delete[] inter;
    delete[] csrRowIdx;

    //}
    CHECK_CUDA(hipEventRecord(stopK));

    // CHECK_CUDA(hipMemcpy(csr_t->row_offsets, d_t_row_ptr, (csr->cols + 1) * sizeof(int), hipMemcpyDeviceToHost));
    // CHECK_CUDA(hipMemcpy(csr_t->col_indices, d_t_col_indices, csr->nnz * sizeof(int), hipMemcpyDeviceToHost));
    // CHECK_CUDA(hipMemcpy(csr_t->values, d_t_values, csr->nnz * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stopK));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipDeviceSynchronize());

    float milliseconds = 0;
    float millisecondsK = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    CHECK_CUDA(hipEventElapsedTime(&millisecondsK, startK, stopK));
    int N = csr->cols;
    float ogbs = (float)(2.0 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / milliseconds;
    float kgbs = (float)(2.0 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / millisecondsK;
    milliseconds /= (float)TRANSPOSITIONS;
    millisecondsK /= (float)TRANSPOSITIONS;
    PRINTF("Time for executing cuCOOt operation: %f ms\n", milliseconds);
    PRINTF("Operation Throughput in GB/s: %7.2f\n", ogbs);

    std::ofstream output;
    output.open("logs/results.log", std::ios::out | std::ios_base::app);
    // algorithm, MatSize, OpTime, Op-GB/s, KTime, K-GB/s
    output << "CSR, " << matrix_size << "x" << matrix_size << ", " <<  milliseconds << ", "<< ogbs << ", " << millisecondsK << ", " << kgbs << "\n"; /* *** */
    output.close();

    if(is_transpose(csr, csr_t)) {
        PRINTF("Transpose is correct.\n");
    } else {
        printf("csr_transposition_2) Transpose is incorrect.\n");
        std::ofstream errlogstream;
        errlogstream.open("logs/transpose_err.log", std::ios::out | std::ios::app);
        errlogstream << "Transpose Error: CSR\n";
        errlogstream << "Original Matrix:\n";
        pretty_print_csr_matrix(csr, errlogstream);
        errlogstream << "\n\nTranposed Matrix:\n";
        pretty_print_csr_matrix(csr_t, errlogstream);
        errlogstream.close();
        PRINTF("--------------------\n");
        return -1;
    }

    CHECK_CUDA(hipFree(d_row_ptr));
    CHECK_CUDA(hipFree(d_col_indices));
    CHECK_CUDA(hipFree(d_values));
    CHECK_CUDA(hipFree(d_t_row_ptr));
    CHECK_CUDA(hipFree(d_t_col_indices));
    CHECK_CUDA(hipFree(d_t_values));
    
    return 0;
}

int csr_transposition(csr_matrix* csr, csr_matrix* csr_t, int matrix_size) {
    PRINTF("--------------------\n");
    PRINTF("CSR Transposition Method Called: csr_transposition() -> transposeCSRToCSC().\n");
    if ((hipSetDevice(0)) != hipSuccess) {
        printf("Failed to set CUDA device\n");
        return 1;
    }
    assert(csr != NULL && csr_t != NULL);
    assert(csr->rows == csr_t->cols && csr->cols == csr_t->rows);
    PRINTF("Transpose CSR to CSC Cuda Method Called: transposeCSRToCSC_cuda().\n");
    if ((hipSetDevice(0)) != hipSuccess) {
        PRINTF("Failed to set CUDA device\n");
        return 1;
    }
    hipEvent_t start, stop;
    hipEvent_t startK1, stopK1;
    hipEvent_t startK2, stopK2;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventCreate(&startK1));
    CHECK_CUDA(hipEventCreate(&stopK1));

    CHECK_CUDA(hipEventCreate(&startK2));
    CHECK_CUDA(hipEventCreate(&stopK2));

    dummy_kernel<<<1,1>>>(); // ? Warm up the GPU

    CHECK_CUDA(hipEventRecord(start));

    // Copy input CSR data to device
    int *d_col_indices, *d_col_counts;
    CHECK_CUDA(hipMalloc((void**)&d_col_indices, csr->nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_col_counts, csr->cols * sizeof(int)));
    CHECK_CUDA(hipMemcpy(d_col_indices, csr->col_indices, csr->nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_col_counts, 0, csr->cols * sizeof(int)));

    int shared_mem_size = 2*(csr->cols) * sizeof(int); //declare the size of the shared memory 
    int *last = new int[1];
    int *d_last;
    CHECK_CUDA(hipMalloc((void**)&d_last, sizeof(int)));

    float *d_values, *d_t_values; //ordered values of trasposed matrix
    int *d_t_col_indices;
    int *d_row_offsets, *d_t_row_offsets;
    CHECK_CUDA(hipMalloc((void**)&d_values, csr->nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_t_values, csr->nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_t_col_indices, csr->nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_row_offsets, (csr->rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_t_row_offsets, (csr->cols + 1) * sizeof(int)));

    int *col_ptr = new int[csr->cols +1];
    int *d_col_ptr;
    CHECK_CUDA(hipMalloc((void**)&d_col_ptr, (csr->cols) * sizeof(int)));
    CHECK_CUDA(hipMemcpy(d_values, csr->values, csr->nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_t_col_indices, 0, csr->nnz * sizeof(int)));

    int *t_col_indices_ordered = new int[csr->nnz];
    int *d_t_col_indices_ordered;
    CHECK_CUDA(hipMalloc((void**)&d_t_col_indices_ordered, csr->nnz * sizeof(int)));

    float millisecondsK = 0;
    int * zeroes = new int[csr->cols];
    memset(zeroes, 0, csr->cols * sizeof(int)); //copy slightly better than memset
    for(int i = 0; i < TRANSPOSITIONS; i++) {
        CHECK_CUDA(hipMemcpy(d_col_counts, zeroes, csr->cols * sizeof(int), hipMemcpyHostToDevice)); //reset col counts to compute correctly

        float millisecondsK1 = 0;
        CHECK_CUDA(hipEventRecord(startK1));
        countNNZPerColumn<<<((csr->nnz + 255) / 256), 256>>>(d_col_indices, d_col_counts, csr->nnz);
        prefix_scan<<<1, (csr->cols), shared_mem_size>>>(d_col_ptr, d_col_counts, csr->cols, d_last);
        CHECK_CUDA(hipEventRecord(stopK1));
        CHECK_CUDA(hipEventSynchronize(stopK1));
        CHECK_CUDA(hipEventElapsedTime(&millisecondsK1, startK1, stopK1));
        //printf("csr) countNNZ + prefix_scan: %f ms\n", millisecondsK1);
        cudaCheckError();
        CHECK_CUDA(hipMemcpy(col_ptr, d_col_ptr, (csr->cols) * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(last, d_last, sizeof(int), hipMemcpyDeviceToHost));
        // https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda : Figure 39-4 
        col_ptr[csr->cols] = last[0];
        csr_t->row_offsets = col_ptr;
        CHECK_CUDA(hipMemcpy(d_row_offsets, col_ptr, (csr->rows + 1) * sizeof(int), hipMemcpyHostToDevice));
        //compute row_offsets in cpu (inclusive_scan)
        int count = 0;
        for(int i = 0; i < csr->cols; i++) {
            int els = csr->row_offsets[i+1] - csr->row_offsets[i];
            for(int j = 0; j < els; j++) {
                csr_t->col_indices[count] = i; //col indices in crescent order
                count++;
            }
        }
        CHECK_CUDA(hipMemcpy(d_t_col_indices, csr_t->col_indices, csr->nnz * sizeof(int), hipMemcpyHostToDevice));
        float millisecondsK2 = 0;
        CHECK_CUDA(hipEventRecord(startK2));
        //order_by_column<<<(csr->cols + 15) /16, 16>>>(d_values, d_col_indices, d_t_values, d_col_ptr, d_col_counts, csr->cols, csr->nnz, d_t_col_indices, d_t_col_indices_ordered);
        order_by_column<<<(csr->cols + 3) /4, 4>>>(d_values, d_col_indices, d_t_values, d_col_ptr, d_col_counts, csr->cols, csr->nnz, d_t_col_indices, d_t_col_indices_ordered);
        CHECK_CUDA(hipEventRecord(stopK2));
        CHECK_CUDA(hipEventSynchronize(stopK2));
        CHECK_CUDA(hipEventElapsedTime(&millisecondsK2, startK2, stopK2));
        //printf("csr) order_by_column: %f ms\n", millisecondsK2);
        millisecondsK += millisecondsK1 + millisecondsK2;
    }
    //return ordered col indices
    CHECK_CUDA(hipMemcpy(csr_t->col_indices, d_t_col_indices_ordered, csr->nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csr_t->values, d_t_values, csr->nnz * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stopK1));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipDeviceSynchronize());

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    int N = csr->cols;
    float ogbs = (float)(2.0 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / milliseconds; 
    float kgbs = (float)(2.0 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / millisecondsK;
    milliseconds /= (float)TRANSPOSITIONS;
    millisecondsK /= (float)TRANSPOSITIONS;
    PRINTF("Time for executing transpose operation: %f ms\n", milliseconds);
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    PRINTF("Operation Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Kernel Throughput in GB/s: %7.2f\n", kgbs);
    std::ofstream output;
    output.open ("logs/results.log", std::ios::out | std::ios_base::app);
    // algorithm, OpTime, Op-GB/s, KTime, K-GB/s
    output << "CSRtoCSCcuda, " << matrix_size << "x" << matrix_size << ", " << milliseconds << ", "<< ogbs << ", " << millisecondsK << ", " << kgbs << "\n"; /* *** */
    output.close();
    cudaCheckError();

    CHECK_CUDA(hipFree(d_col_indices));
    CHECK_CUDA(hipFree(d_col_counts));
    CHECK_CUDA(hipFree(d_col_ptr));
    CHECK_CUDA(hipFree(d_last));
    CHECK_CUDA(hipFree(d_values));
    CHECK_CUDA(hipFree(d_t_values));
    CHECK_CUDA(hipFree(d_t_col_indices));
    CHECK_CUDA(hipFree(d_row_offsets));
    CHECK_CUDA(hipFree(d_t_row_offsets));
    CHECK_CUDA(hipFree(d_t_col_indices_ordered));

    PRINTF("Transpose Completed.\n");

    if (is_transpose(csr, csr_t)) {
        PRINTF("Transpose is correct.\n");
    } else {
        printf("csr_transposition) Transpose is incorrect.\n");
        std::ofstream errlogstream;
        errlogstream.open("logs/transpose_err.log", std::ios::out | std::ios::app);
        errlogstream << "Transpose Error: CSR to CSC\n";
        errlogstream << "Original Matrix:\n";
        pretty_print_csr_matrix(csr, errlogstream);
        errlogstream << "\n\nTranposed Matrix:\n";
        pretty_print_csr_matrix(csr_t, errlogstream);
        errlogstream.close();
        PRINTF("--------------------\n");
        return -1;
    }
    PRINTF("--------------------\n");
    return 0;
}

int block_trasposition(float* mat, unsigned int N, int matrix_size) {
    PRINTF("--------------------\n");
    PRINTF("Block Transposition Method Called: block_transposition() -> block_transpose().\n");
    if ((hipSetDevice(0)) != hipSuccess) {
        printf("Failed to set CUDA device\n");
        return 1;
    }
    hipEvent_t start, stop;
    hipEvent_t startK, stopK;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventCreate(&startK));
    CHECK_CUDA(hipEventCreate(&stopK));

    dummy_kernel<<<1,1>>>(); // ? Warm up the GPU

    CHECK_CUDA(hipEventRecord(start));

    int mem_size = N * N * sizeof(float);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    initMatrix(mat, N);
    float *d_mat, *d_mat_t;
    //int threads, blocks = 0;
    PRINTF("Allocating memory.\n");
    CHECK_CUDA(hipMalloc((void**)&d_mat, mem_size));
    CHECK_CUDA(hipMalloc((void**)&d_mat_t, mem_size));
    //hipMalloc((void**)&d_mat_t, mem_size);
    PRINTF("Memory allocated.\n");
    //copy data to gpu

    PRINTF("Data copied.\n");
    //setup grid and block size
    dim3 DimGrid = {N/TILE_SIZE, N/TILE_SIZE, 1};
    dim3 DimBlock = {TILE_SIZE, BLOCK_ROWS, 1};

    CHECK_CUDA(hipMemcpy(d_mat, mat, mem_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipEventRecord(startK));
    for(int i = 0; i < TRANSPOSITIONS; i++){
        block_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t, N);
    }
    CHECK_CUDA(hipEventRecord(stopK));
    PRINTF("Kernel returned\n");

    //copy data back
    CHECK_CUDA(hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost));
    //sync
    CHECK_CUDA(hipEventRecord(stop));

    CHECK_CUDA(hipEventSynchronize(stopK));
    CHECK_CUDA(hipEventSynchronize(stop));

    CHECK_CUDA(hipDeviceSynchronize());

    float millisecondsK = 0;
    CHECK_CUDA(hipEventElapsedTime(&millisecondsK, startK, stopK));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    float ogbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / milliseconds;
    float kgbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / millisecondsK;
    milliseconds /= (float)TRANSPOSITIONS;
    millisecondsK /= (float)TRANSPOSITIONS;
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    PRINTF("Operation Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Kernel Throughput in GB/s: %7.2f\n", kgbs);

    std::ofstream output;
    output.open("logs/results.log", std::ios::out | std::ios_base::app);

    // algorithm, OpTime, Op-GB/s, KTime, K-GB/s
    output << "block, " << matrix_size << "x" << matrix_size << ", " << milliseconds << ", "<< ogbs << ", " << millisecondsK << ", " << kgbs << "\n";
    output.close();

    CHECK_CUDA(hipEventDestroy(startK));
    CHECK_CUDA(hipEventDestroy(stopK));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    //test if the matrix is transposed
    int ret = 0;
    if (testTranspose(mat, mat_t, N) != 0) {
        printf("block_transpose) Transpose is incorrect.\n");
        // save the matrix to a file
        std::ofstream output;
        output.open("logs/block_transpose_err.log", std::ios::out | std::ios_base::app);
        output << "Matrix: " << N << "x" << N << "\n";
        output << "Original Matrix:\n";
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                output << mat[i + j*N] << " ";
            }
            output << "\n";
        }
        output << "\n\nTransposed Matrix:\n";
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                output << mat_t[i + j*N] << " ";
            }
            output << "\n";
        }
        output.close();
        ret = -1;
    }
    PRINTF("--------------------\n");
    //free gpu resources
    CHECK_CUDA(hipFree(d_mat));
    CHECK_CUDA(hipFree(d_mat_t));
    free(mat_t);
    return ret;
}

int conflict_transposition(float* mat, unsigned int N, int matrix_size) {
    PRINTF("--------------------\n");
    PRINTF("Conflict Transposition Method Called: conflict_transposition() -> conflict_transpose().\n");
    if ((hipSetDevice(0)) != hipSuccess) {
        printf("Failed to set CUDA device\n");
        return 1;
    }
    hipEvent_t start, stop;
    hipEvent_t startK, stopK;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventCreate(&startK));
    CHECK_CUDA(hipEventCreate(&stopK));

    dummy_kernel<<<1,1>>>(); // ? Warm up the GPU

    CHECK_CUDA(hipEventRecord(start));
    PRINTF("Cuda Events Created.\n");

    int mem_size = N * N * sizeof(float);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    float *d_mat, *d_mat_t;
    //int threads, blocks = 0;
    PRINTF("Allocating memory.\n");
    CHECK_CUDA(hipMalloc((void**)&d_mat, mem_size));
    CHECK_CUDA(hipMalloc((void**)&d_mat_t, mem_size));
    PRINTF("Memory allocated.\n");
    PRINTF("Now copying data from host (mat) to device (d_mat). Exactly: %d Bytes\n", mem_size);
    PRINTF("Data copied.\n");
    //setup grid and block size
    dim3 DimGrid = {N/TILE_SIZE, N/TILE_SIZE, 1};
    dim3 DimBlock = {TILE_SIZE, BLOCK_ROWS, 1};
    
    CHECK_CUDA(hipMemcpy(d_mat, mat, mem_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipEventRecord(startK));
    for(int i = 0; i < TRANSPOSITIONS; i++){
        conflict_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    }
    CHECK_CUDA(hipEventRecord(stopK));
    PRINTF("Kernel returned\n");

    //copy data back
    CHECK_CUDA(hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost));
    //sync
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stopK));
    CHECK_CUDA(hipEventSynchronize(stop));

    CHECK_CUDA(hipDeviceSynchronize());
    
    float millisecondsK = 0;
    CHECK_CUDA(hipEventElapsedTime(&millisecondsK, startK, stopK));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    float ogbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / milliseconds;
    float kgbs = (float)(2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS) / millisecondsK;
    milliseconds /= (float)TRANSPOSITIONS;
    millisecondsK /= (float)TRANSPOSITIONS;
    PRINTF("Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Throughput in GB/s: %7.2f\n", kgbs);

    std::ofstream output;
    output.open("logs/results.log", std::ios::out | std::ios_base::app);
    // algorithm, OpTime, Op-GB/s, KTime, K-GB/s
    output << "Conflict, " << matrix_size << "x" << matrix_size << ", " <<  milliseconds << ", "<< ogbs << ", " << millisecondsK << ", " << kgbs << "\n";
    output.close();

    CHECK_CUDA(hipEventDestroy(startK));
    CHECK_CUDA(hipEventDestroy(stopK));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    //test if the matrix is transposed
    int ret =  testTranspose(mat, mat_t, N);
    if (ret != 0) {
        printf("conflict_transpose) Transpose is incorrect.\n");
    }
    PRINTF("--------------------\n");
    //free gpu resources
    CHECK_CUDA(hipFree(d_mat));
    CHECK_CUDA(hipFree(d_mat_t));
    free(mat_t);
    return ret;
}
