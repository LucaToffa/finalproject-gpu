#include "benchmarks.h"
#include "commons.h"
#include "defines.h"
#include <hip/hip_runtime.h>
#include "kernels.h"

//instead of the normal initMatrix, each benchmark should be loaded with a matrix from ./matrices/

int block_benchmark(uint N){
    //give access to the gpu
    int mem_size = N * N * sizeof(float);
    float* mat = (float*) malloc(mem_size);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    initMatrix(mat, N);
    float *d_mat, *d_mat_t;
    hipError_t err;
    //int threads, blocks = 0;
    PRINTF("Allocating memory\n");
    if((err = hipMalloc((void**)&d_mat, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_a: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    if((err = hipMalloc((void**)&d_mat_t, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //hipMalloc((void**)&d_mat_t, mem_size);
    PRINTF("Memory allocated\n");
    //copy data to gpu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if((err = hipMemcpy(d_mat, mat, N * N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess){
        printf("Error copying data to d_mat: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    PRINTF("Data copied\n");
    //setup grid and block size
    dim3 DimGrid = {N/TILE_SIZE, N/TILE_SIZE, 1};
    dim3 DimBlock = {TILE_SIZE, BLOCK_ROWS, 1};
    
    //call kernel as many times as needed
    //first a dummy kernel
    block_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    hipEvent_t startK, stopK;
    hipEventCreate(&startK);
    hipEventCreate(&stopK);
    hipEventRecord(startK);
    for(int i = 0; i < TRANSPOSITIONS; i++){
        block_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    }
    hipEventRecord(stopK);
    hipEventSynchronize(stopK);
    PRINTF("Kernel returned\n");

    //copy data back
    if((err = hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost)) != hipSuccess){
        printf("Error copying data to mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //sync
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    
    float millisecondsK = 0;
    hipEventElapsedTime(&millisecondsK, startK, stopK);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float ogbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / milliseconds;
    float kgbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / millisecondsK;
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    PRINTF("Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Throughput in GB/s: %7.2f\n", kgbs);
    printf("%f, %f, %f, %f\n", milliseconds, ogbs, millisecondsK, kgbs);
    hipEventDestroy(startK);
    hipEventDestroy(stopK);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    //results
    printMatrix(mat_t, N);

    //test if the matrix is transposed
    testTranspose(mat, mat_t, N);  

    //free gpu resources
    hipFree(d_mat);
    hipFree(d_mat_t);
    free(mat);
    free(mat_t);
    return 0;
}

int basic_benchmark(uint N){
    //give access to the gpu
    int mem_size = N * N * sizeof(float);
    float* mat = (float*) malloc(mem_size);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    initMatrix(mat, N);
    float *d_mat, *d_mat_t;
    hipError_t err;
    //int threads, blocks = 0;
    PRINTF("Allocating memory\n");
    if((err = hipMalloc((void**)&d_mat, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_a: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    if((err = hipMalloc((void**)&d_mat_t, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //hipMalloc((void**)&d_mat_t, mem_size);
    PRINTF("Memory allocated\n");

    //copy data to gpu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if((err = hipMemcpy(d_mat, mat, mem_size, hipMemcpyHostToDevice)) != hipSuccess){
        printf("Error copying data to d_mat: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    PRINTF("Data copied\n");
    //setup grid and block size
    dim3 gridB(N / B_TILE, N / B_TILE);
    dim3 blockB(B_TILE, B_ROWS);
    
    //call kernel as many times as needed
    //first a dummy kernel
    basic_transpose<<<gridB, blockB>>>(d_mat, d_mat_t, N);
    hipEvent_t startK, stopK;
    hipEventCreate(&startK);
    hipEventCreate(&stopK);
    hipEventRecord(startK);
    for(int i = 0; i < TRANSPOSITIONS; i++){
        basic_transpose<<<gridB, blockB>>>(d_mat, d_mat_t, N);
    }
    hipEventRecord(stopK);
    hipEventSynchronize(stopK);
    PRINTF("Kernel returned\n");

    //copy data back
    if((err = hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost)) != hipSuccess){
        printf("Error copying data to mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //sync
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    
    float millisecondsK = 0;
    hipEventElapsedTime(&millisecondsK, startK, stopK);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float ogbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / milliseconds;
    float kgbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / millisecondsK;
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    PRINTF("Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Throughput in GB/s: %7.2f\n", kgbs);
    printf("%f, %f, %f, %f, ", milliseconds, ogbs, millisecondsK, kgbs);

    hipEventDestroy(startK);
    hipEventDestroy(stopK);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //test if the matrix is transposed
    PRINTF("basic results:\n");
    printMatrix(mat_t, N);
    testTranspose(mat, mat_t, N);  

    //reset output matrix
    memset(mat_t, 0, mem_size);
    //hipMemset(d_mat_t, 0, mem_size);
    hipFree(d_mat);
    hipFree(d_mat_t);
    free(mat);
    free(mat_t);
    return 0;
}

int conflict_benchmark(uint N){
    //give access to the gpu
    int mem_size = N * N * sizeof(float);
    float* mat = (float*) malloc(mem_size);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    initMatrix(mat, N);
    float *d_mat, *d_mat_t;
    hipError_t err;
    //int threads, blocks = 0;
    PRINTF("Allocating memory\n");
    if((err = hipMalloc((void**)&d_mat, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_a: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    if((err = hipMalloc((void**)&d_mat_t, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //hipMalloc((void**)&d_mat_t, mem_size);
    PRINTF("Memory allocated\n");
    //copy data to gpu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if((err = hipMemcpy(d_mat, mat, N * N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess){
        printf("Error copying data to d_mat: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    PRINTF("Data copied\n");
    //setup grid and block size
    dim3 DimGrid = {N/TILE_SIZE, N/TILE_SIZE, 1};
    dim3 DimBlock = {TILE_SIZE, BLOCK_ROWS, 1};
    
    //call kernel as many times as needed
    //first a dummy kernel
    conflict_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    hipEvent_t startK, stopK;
    hipEventCreate(&startK);
    hipEventCreate(&stopK);
    hipEventRecord(startK);
    for(int i = 0; i < TRANSPOSITIONS; i++){
        conflict_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    }
    hipEventRecord(stopK);
    hipEventSynchronize(stopK);
    PRINTF("Kernel returned\n");

    //copy data back
    if((err = hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost)) != hipSuccess){
        printf("Error copying data to mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //sync
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    
    float millisecondsK = 0;
    hipEventElapsedTime(&millisecondsK, startK, stopK);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    float ogbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / milliseconds;
    float kgbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / millisecondsK;
    PRINTF("Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Throughput in GB/s: %7.2f\n", kgbs);
    printf("%f, %f, %f, %f, ", milliseconds, ogbs, millisecondsK, kgbs);

    hipEventDestroy(startK);
    hipEventDestroy(stopK);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    //results
    printMatrix(mat_t, N);

    //test if the matrix is transposed
    testTranspose(mat, mat_t, N);  

    //free gpu resources
    hipFree(d_mat);
    hipFree(d_mat_t);
    free(mat);
    free(mat_t);
    return 0;
}